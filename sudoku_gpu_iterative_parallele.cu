#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <stack>
#include <iterator>
#include <algorithm>
#include <numeric>
using namespace std;
#include <getopt.h>
#include "grid.h"


#include "gpu_grid.cu"

// ==================================================================
// CUDA / CUME
// ==================================================================
#include <hip/hip_runtime.h>

#define H2D hipMemcpyHostToDevice
#define D2H hipMemcpyDeviceToHost

// ------------------------------------------------------------------
// definition of a macro instruction that checks if a CUDA function
// was successull or not. If the execution of the function resulted
// in some error we display it and stop the program
// ------------------------------------------------------------------
#define cume_check(value) {     \
  hipError_t err = value; \
  if (err != hipSuccess) {       \
    cerr << endl; \
    cerr << "============================================\n"; \
    cerr << "Error: " << hipGetErrorString(err) << " at line "; \
    cerr << __LINE__ << " in file " <<  __FILE__;   \
    cerr <<  endl; \
    exit(EXIT_FAILURE); \
  } \
}
 
// ------------------------------------------------------------------
// Same as cuda_check but for kernel. This macro instruction is used
// after the execution of the kernel (see the macros KERNEL_EXECUTE_NR
// and KERNEL_EXECUTE_WR in cume_kernel.h)
// ------------------------------------------------------------------
#define cume_check_kernel() { \
  hipError_t err = hipGetLastError(); \
  if (err != hipSuccess)  { \
    cerr << endl; \
    cerr << "============================================\n"; \
    cerr << "Kernel Error: " << hipGetErrorString(err) << " at line "; \
    cerr << __LINE__ << " in file " <<  __FILE__;   \
    cerr <<  endl; \
    exit(EXIT_FAILURE); \
  } \
}

#define dump(var) cout << #var << "=" << var << endl;

ostream& operator<<( ostream& out, Position& pos ) {
	return Position_print( out, pos );
}

ostream& operator<<( ostream& out, PositionCost& pc ) {
	return PositionCost_print( out, pc );
}

ostream& operator<<( ostream& out, BlockCost& bc ) {
	return BlockCost_print( out, bc );
}

ostream& operator<<( ostream& out, Grid& grid ) {
	return Grid_print( out, grid );
}

// ==================================================================
// GLOBAL VARIABLES
// ==================================================================
int nbr_solutions = 0;
int verbose_level = 1;
bool reverse_flag = false;
int nbr_blocks = 1;
bool print_first_flag = false;

string satisfied_strings[] = { 
	"unsatisfied", 
	"almost", 
	"*** not defined ***", 
	"satisfied" 
};

extern __device__ int GPU_Grid_satisfied( Grid *g );

/**
 * Iteratively solve the Sudoku given the list of zero positions
 *
 */
__device__
int device_Grid_solve_iterative( int gtid, 
	Grid& g, 
	int *tab_nbr_solutions,
	int nbr_positions, 
	Position* tab_positions ) {
	
	int nb_sol = 0;
	 
	int m = nbr_positions; 
	
	int i = 0;
	while (i < m) {
				
		if (i == m) {
		
			--i;
			
		} else {	
			
			int y = tab_positions[ i ]._y;
			int x = tab_positions[ i ]._x;
			++g[ y ][ x ];
			
			
			bool exit_loop = false;
			while ((i > 0) and (exit_loop == false)) {
				if ( g[ y ][ x ] > MAX_VAL) {
					g[ y ][ x ] = 0;
					--i;
					y = tab_positions[ i ]._y;
					x = tab_positions[ i ]._x;
			
					++g[ y ][ x ];
				} else {
					exit_loop = true;
				}
			}
		}

		if ( g[ tab_positions[ 0 ]._y ][ tab_positions[ 0 ]._x ] > MAX_VAL) break;
		
		int sat = GPU_Grid_satisfied( g );
		
		
		if ( sat  != UNSATISFIED ) {
			++i;
		}
				
		if (i == m) {
			
			if ( GPU_Grid_satisfied( g ) == SATISFIED ) {
					
				++nb_sol;

			}
			
			--i;		
		}	
		
	}
	
	return nb_sol;

}

__global__
void kernel_Grid_solve_iterative( int nbr_grids, 
	Grid *tab_grids, 
	int *tab_nbr_solutions,
	int nbr_positions, 
	Position* tab_positions ) {


	int gtid = blockDim.x * blockIdx.x + threadIdx.x;
		
	if (gtid < nbr_grids) {
	
		tab_nbr_solutions[ gtid ] =
		device_Grid_solve_iterative( gtid, 
			tab_grids[ gtid ],
			tab_nbr_solutions,
			nbr_positions, 
			tab_positions 
		);
		
	}
	
	
}

/**
 * Iteratively solve the Sudoku given the list of zero positions
 *
 */
void Grid_solve_iterative( int nbr_grids, Grid *cpu_tab_grids ) {

	vector< Position > empty_positions;
	
	Grid_find_empty_positions( cpu_tab_grids[ 0 ], empty_positions );

	int nbr_positions = static_cast<int>( empty_positions.size() );
	
	if (reverse_flag) {
	
		reverse( empty_positions.begin(), empty_positions.end() );
		
	}
	
	if (verbose_level >= 2) {
		cout << "- empty positions=" << empty_positions.size() << endl;
		for (auto p : empty_positions) {
			cout << p << endl;
		}
	}

	Position *cpu_tab_positions = new Position[ nbr_positions ] ;
	{
	
		int i = 0;
		for (auto p : empty_positions) {
			cpu_tab_positions[ i ] = p;
			++i;
		}
		
	}
	
	Position *gpu_tab_positions;

	cume_check( hipMalloc( (void **) &gpu_tab_positions, nbr_positions * sizeof( Position ) ) );
	cume_check( hipMemcpy( gpu_tab_positions, cpu_tab_positions, nbr_positions * sizeof( Position ), H2D ) );

	int *cpu_tab_nbr_solutions = new int [ nbr_grids ];

	int *gpu_tab_nbr_solutions;
	
	cume_check( hipMalloc( (void **) &gpu_tab_nbr_solutions, nbr_grids * sizeof( int ) ) );
	//cume_check( hipMemset( gpu_tab_nbr_solutions, 0, nbr_grids * sizeof( int ) ) );


	Grid *gpu_tab_grids;
	
	cume_check( hipMalloc( (void **) &gpu_tab_grids, nbr_grids * sizeof(Grid) ) );
	cume_check( hipMemcpy( gpu_tab_grids, cpu_tab_grids, nbr_grids * sizeof(Grid), H2D ) );

	dim3 cuda_grid(1,1,1) , cuda_block(1,1,1);
	
	const int MAX_THREADS_PER_BLOCK = 1024;
	
	if (nbr_grids < MAX_THREADS_PER_BLOCK) {
		cuda_block.x = nbr_grids;
	} else {
		cuda_block.x = MAX_THREADS_PER_BLOCK;
		cuda_grid.x = ((nbr_grids + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK);
	}
	
	cout << endl;
	cout << "- cuda  grid( x=" << cuda_grid.x << ",y=" << cuda_grid.y << ",z=" << cuda_grid.z << " )" << endl;
	cout << "- cuda block( x=" << cuda_block.x << ",y=" << cuda_block.y << ",z=" << cuda_block.z << " )" << endl;
	
	cout << "- start kernel" << endl;
	
	kernel_Grid_solve_iterative<<< cuda_grid, cuda_block >>>( nbr_grids,
		gpu_tab_grids, 
		gpu_tab_nbr_solutions, 
		nbr_positions,
		gpu_tab_positions 
	);
	cume_check_kernel();
	
	cume_check( hipMemcpy( cpu_tab_grids, gpu_tab_grids, nbr_grids * sizeof(Grid), D2H) );
	
	cume_check( hipMemcpy( cpu_tab_nbr_solutions, gpu_tab_nbr_solutions, nbr_grids * sizeof(int), D2H) );
	
	//cout << "solutions:" << endl;
	//copy( &cpu_tab_nbr_solutions[ 0 ], &cpu_tab_nbr_solutions[ nbr_grids ], ostream_iterator<int>( cout, " ") ); 
	
	nbr_solutions = std::accumulate( &cpu_tab_nbr_solutions[ 0 ], &cpu_tab_nbr_solutions[ nbr_grids ], 0 );
	
	hipFree( gpu_tab_positions );
	hipFree( gpu_tab_grids );
	hipFree( gpu_tab_nbr_solutions );
	
	delete [] cpu_tab_nbr_solutions;
}


/**
 * Recursively instantiate some cells
 *
 */
void fill_grids( Grid *tab_grids, int nbr_blocks, 
		vector< BlockCost >& blocks_costs, int& grid_id, int n ) {
	
	if ( n >= nbr_blocks ) {
	
		for (int i = 0; i < n; ++i) {

			int block = blocks_costs[ i ]._block;
			vector<int>& values = blocks_costs[ i ]._values;
			
			Grid_fill_block( tab_grids[ grid_id ], block, values );
			
		}
		
		++grid_id;
		
	} else {
	
		vector<int>& values = blocks_costs[ n ]._values;
		
		do {
		
			fill_grids( tab_grids, nbr_blocks, blocks_costs, grid_id, n + 1 );
			
		} while ( next_permutation( values.begin(), values.end() ) );
		
	}
	
}


/**
 * main function
 *
 */
int main( int argc, char *argv[] ) {

	string input;
	string input_file_name;
		
	static struct option long_options[] = {
	
		{ "verbose-level", required_argument, 0,  0 },
		{ "input", required_argument, 0,  0 }, 
		{ "blocks", required_argument, 0,  0 }, 
		{ "reverse", no_argument, 0, 0 }, 
		{ "print-first", no_argument, 0, 0 },
		{ 0, 0, 0, 0 }
		
	};

	int option_index = 0;
	while (true) {
	
		int c = getopt_long( argc, argv, "v:i:b:rf", long_options, &option_index );
	
		if (c == -1) break;

		switch( c ) {
			case 'v': 
				verbose_level = atoi( optarg );
				break;
				
			case 'i':
				input_file_name = optarg;
				break;
					
			case 'r':
				reverse_flag = true;
				break;
				
			case 'b':
				nbr_blocks = atoi( optarg );
				break;	
			
			case 'f': 
				print_first_flag = true;
				break;
				
			default:
				cerr << "Unknown option	!" << endl;
				exit( EXIT_FAILURE );
		}
		
	}
		
	srand( time( nullptr ) );
	
	Grid initial_grid;
	
	Grid_init( initial_grid );
		
	// 2315 solutions
	input = 
	"1 2 5 7 3 6 4 0 0 "
	"0 0 0 0 1 8 5 0 0 "
	"0 0 0 5 0 4 0 0 0 "
	"0 0 0 0 5 0 0 2 0 "
	"0 0 0 0 0 0 0 0 0 "
	"0 0 0 2 7 0 0 0 0 "
	"0 8 3 0 0 0 0 0 0 "
	"0 0 0 6 0 2 3 9 0 "
	"0 0 0 0 0 0 8 4 7 ";
	

	
	if (input_file_name.size() != 0) {
		
		cout << "- read file " << input_file_name << endl;

		ifstream ifs( input_file_name );
				
		if (ifs.is_open()) {

			std::string str( (std::istreambuf_iterator<char>(ifs)),
                 std::istreambuf_iterator<char>());
                 
            input = str;     

		} else {
		
			cout << "! error: could not open file '" << input_file_name << "'" << endl;
			
		}
	}
	
	Grid_fill( initial_grid, input );
	
	if (verbose_level >= 1) {
		cout << endl;
		cout << "- initial grid" << endl;
		cout << initial_grid << endl;
	}
		
	
	int satisfiability = Grid_satisfied( initial_grid );
	cout << endl;
	cout << "- initial grid satisfied ? " << satisfied_strings[ satisfiability ] << endl;
	cout << endl;
	
	if (satisfiability == UNSATISFIED) {
	
		cout << "! error: can't solve a grid that is already UNSATISFIED" << endl;
			
	} else {
	
		vector< BlockCost > blocks_costs;
		
		for (int block = MIN_VAL; block <= MAX_VAL; ++block) {
			BlockCost bc;
			bc._block = block;
			bc._permutations = 0;
			bc._cost = Grid_block_nbr_values_set( initial_grid, block, bc._values );
			if (Grid_blk_satisfied( initial_grid, block ) != SATISFIED) {
			
				blocks_costs.push_back( bc );
				
			}
		}
		
		// sort by most constrained block
		std::sort( blocks_costs.begin(), blocks_costs.end(), []( BlockCost& a, BlockCost& b) {
			return a._cost > b._cost;
		} );	
		
		//
		// By using 'nbr_blocks' and their permutations, 
		// we estimate the number of grids to use and fill
		//
				
		cout << "- evaluate number of permutations:" << endl;
				
		int total_permutations = 1;
		
		for (int i = 0; i < nbr_blocks; ++i) {
		
			int nbr_permutations = 0;		
			
			std::vector<int>& values = blocks_costs[ i ]._values;
			
			do {
			
				++nbr_permutations;
				
			} while (next_permutation( values.begin(), values.end()));
		
			if (verbose_level >= 1) {
				cout << "-- number of permutations for block " << blocks_costs[ i ]._block;
				cout << " = " << nbr_permutations << endl;		
			}
			
			blocks_costs[ i ]._permutations = nbr_permutations;
			
			total_permutations *= nbr_permutations;
		}
		
		if (verbose_level >= 2) {
			cout << "blocks costs=" << blocks_costs.size() << endl;
			for (auto bc : blocks_costs) {
				cout << bc << endl;
			}
		}
		
		cout << "- total number of permutations with " << nbr_blocks << " block(s) = ";
		cout << total_permutations << endl;;
		
	
		//
		// Now we create the grids from the initial grid and fill them
		//
		int nbr_grids = total_permutations;
		
		Grid *tab_grids = new Grid[ nbr_grids ];
		
		for (int i = 0; i < nbr_grids; ++i) {
		
			Grid_copy( tab_grids[ i ], initial_grid );
			
		}
		
		int grid_id = 0;
		
		fill_grids( tab_grids, nbr_blocks, blocks_costs, grid_id, 0 );
		
		if (verbose_level >= 2) {

			cout << "- all grids" << endl;
			
			for (int i = 0; i < nbr_grids; ++i) {
			
				cout << i << endl << tab_grids[ i ] << endl;
			}
		
		}	
				
		Grid_solve_iterative( nbr_grids, tab_grids );
		
	}	
	
	cout << endl;	
	cout << "- number of solutions=" << nbr_solutions << endl;
		
	return EXIT_SUCCESS;
}
	
